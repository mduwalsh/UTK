
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


int blockSize=256; 
int gridSize=256;

__global__ void gameOfLife(int *indata, int *outdata, int width, int height) 
{
	__shared__ int sodata[2566];
	__shared__ int sidata[256*3];
	int tSize=width*height;
	int x, y, x0,x1,y0,y1, n;
	int tid;
	int bid;
	int cid;
	for(bid=0;bid<gridDim.x;bid++)
	{
		tid=threadIdx.x;
		cid=bid*blockDim.x+threadIdx.x;
		if(tid<width)
		{
			
			y=(blockIdx.x*blockDim.x+threadIdx.x) / width; // y position
			y0=(y-1+height) % height; // one row up y positional value
			y1=(y+1) % height; // one row below y positional value
			// load into shared memory
			sidata[tid]=indata[y0*width+tid]; // upper row
			sidata[width+tid]=indata[y*width+tid]; // computing cell containing row
			sidata[2*width+tid]=indata[y1*width+tid]; // lower row		
				
		}
		__syncthreads();
		
	}

	for(bid=0;bid<height;bid++)
	{
		if(threadIdx.x<width)
		{
			x=(bid*blockDim.x+threadIdx.x) % width; // x position
			x0=(x-1+width) % width; // one left x positional value assuming circular edge connecting every cell in edges too
			x1=(x+1) % width; // one right x positional value
			
			n=sidata[2*width+x0]+sidata[width+x0]+sidata[x0]+sidata[x]+sidata[x1]+sidata[width+x1]+sidata[2*width+x1]+sidata[2*width+x]; // no. of alive neighbor cells
			if(n==3 || (n==2 && sidata[width+x]==1))
			{
				sodata[threadIdx.x]=1;
			}
			else
			{
				sodata[threadIdx.x]=0;
			}
			
		}	
		__syncthreads();
		//cid+=gridDim.x*blockDim.x;	
	}
	__syncthreads();
	outdata[cid]=sodata[tid];
	__syncthreads();
}

int main()
{
	clock_t sTime=clock();
	
	int width=256;
	int height=256;
	int dsize=width*height;	
	int iteration=1;
	int i, j;

	int *data;
	int *d_indata, *d_outdata, *temp;

	// allocate memory for data in host 
	data=(int *)malloc(dsize*sizeof(int));
	// allocate memory for data in device
	hipMalloc(&d_indata, dsize*sizeof(int));
	hipMalloc(&d_outdata, dsize*sizeof(int));

	// initialize data in host as randomly 0 or 1
	for(i=0;i<dsize;i++)
	{
		data[i]=rand()%2;
	}

	// diplay cell status in console
	printf("\n board status # \n");
	for(i=0;i<height;i++)
	{
		for(j=0;j<width;j++)
		{
			printf(" %d",data[i*width+j]);
		}
		printf("\n");
	}

	// copy initialized data to gpu device
	hipMemcpy( d_indata, data, dsize, hipMemcpyHostToDevice ); 

	for(i=0;i<iteration;i++)
	{
		// call kernel
		gameOfLife<<<gridSize, blockSize>>>(d_indata, d_outdata, width, height);
		// synchronize between thread blocks
		hipDeviceSynchronize();
		// swap d_indata and d_outdata for next iteration
		temp=d_indata;
		d_indata=d_outdata;
		d_outdata=temp;
	}

	// copy data back from device to host memory	
	hipMemcpy( data, d_indata, dsize, hipMemcpyDeviceToHost); 

	// diplay cell status in console
	printf("\n board status # \n");
	for(i=0;i<height;i++)
	{
		for(j=0;j<width;j++)
		{
			printf(" %d",data[i*width+j]);
		}
		printf("\n");
	}
	
	// free memory allocated
	hipFree(d_indata);
	hipFree(d_outdata);
	free(data);
	clock_t eTime=clock();
	printf("time taken = %ld",eTime-sTime);
	hipDeviceReset();
	return 0;
}
