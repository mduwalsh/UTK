
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

int blockSize; 
int gridSize;

__global__ void gameOfLife(int *indata, int *outdata, int width, int height) 
{
	__shared__ int sdata[256];
	int tSize=width*height;
	int x, y, x0,x1,y0,y1, n;
	int bid, cid, tid;
	tid = threadIdx.x;
	bid = blockIdx.x;
	for(cid = blockIdx.x*blockDim.x+tid; cid < tSize; cid += gridDim.x*blockDim.x){
	  if(tid<width){
		  x=(cid) % width; // x position
		  y=(cid) / width; // y position
		  x0=(x-1+width) % width; // one left x positional value assuming circular edge connecting every cell in edges too
		  x1=(x+1) % width; // one right x positional value
		  y0=(y-1+height) % height; // one row up y positional value
		  y1=(y+1) % height; // one row below y positional value
		  n=indata[y1*width+x0]+indata[y*width+x0]+indata[y0+x0]+indata[y0*width+x]+indata[y0*width+x1]+indata[y*width+x1]+indata[y1*width+x1]+indata[y1*width+x]; // no. of alive neighbor cells
		  
		  if(n==3 || (n==2 && indata[cid]))  // life created or sustains
		  {
			  sdata[tid]=1;
			  //outdata[cid]=1;
		  }
		  else  // life destroyed or stay died
		  {
			  sdata[tid]=0;
			  //outdata[cid]=0;
		  }						
		  __syncthreads();
		  outdata[cid]=sdata[tid];
	  }	  	  
	}
}

int main(int argc, char **argv)
{
	if(argc ^ 4){
	  printf("Usage: ./gol seed height width iteration\n");
	  exit(1);
	}
	clock_t sTime=clock();
	int width= atoi(argv[3]);
	int height=atoi(argv[2]);
	int dsize=width*height;	
	int iteration=atoi(argv[3]);
	int i, j;

	int *data;
	int *d_indata, *d_outdata, *temp;

	// allocate memory for data in host 
	data=(int *)malloc(dsize*sizeof(int));
	
	// allocate memory for data in device
	hipMalloc(&d_indata, dsize*sizeof(int));
	hipMalloc(&d_outdata, dsize*sizeof(int));
	
	srand(atoi(argv[0]));                                   // test seed
	// initialize data in host as randomly 0 or 1
	for(i=0;i<dsize;i++)
	{
		data[i]=rand()%2;
	}

	// copy initialized data to gpu device
	hipMemcpy( d_indata, data, dsize, hipMemcpyHostToDevice ); 

	for(i=0;i<iteration;i++)
	{
		// call kernel
		gameOfLife<<<gridSize, blockSize>>>(d_indata, d_outdata, width, height);
		// synchronize between thread blocks
		hipDeviceSynchronize();
		// swap d_indata and d_outdata for next iteration
		temp=d_indata;
		d_indata=d_outdata;
		d_outdata=temp;
	}

	// copy data back from device to host memory	
	hipMemcpy( data, d_indata, dsize, hipMemcpyDeviceToHost); 
	
	// diplay cell status in console
	printf("\n board status # \n");
	for(i=0;i<height;i++)
	{
		for(j=0;j<width;j++)
		{
			printf(" %d",data[i*width+j]);
		}
		printf("\n");
	}
	
	// free memory allocated
	hipFree(d_indata);
	hipFree(d_outdata);
	free(data);
	clock_t eTime=clock();
	printf("time taken = %ld",eTime-sTime);
	hipDeviceReset();
	return 0;
}
